#include "hip/hip_runtime.h"
#include "lenet.h"

/* ----- OTHERS FUNCTIONS ----- */
number ReLU(number x) {
    return x*(x > 0);
}

number ReLU_GRAD(number x) {
    return x > 0;
}

void softMax(Feature *input, uint8 target, Feature *featureGradient) {
    //int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tn = threadIdx.y, tm = threadIdx.x;
    //int tid  = threadIdx.x;

    // do reduction in global mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            d_in[myId] += d_in[myId + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = d_in[myId];
    }
    //Aux variables
    uint8 on, om;
    number den = 0, inner = 0;
    Matrix *inputMatrix = FEATURE_GETMATRIX(input, 0);
    Matrix *gradientMatrix = FEATURE_GETMATRIX(featureGradient, 0);
    //Error and softmax
    for(on = 0; on < inputMatrix->m; on++) {
        den = 0;
        for(om = 0; om < inputMatrix->m; om++)
            den += exp(MATRIX_VALUE1(inputMatrix, om) - MATRIX_VALUE1(inputMatrix, on));
        ARRAY_VALUE(gradientMatrix, on) = 1.0/den;
        inner -= ARRAY_VALUE(gradientMatrix, on) * ARRAY_VALUE(gradientMatrix, on);
    }
    inner += ARRAY_VALUE(gradientMatrix, target);
    for(om = 0; om < gradientMatrix->m; om++)
        ARRAY_VALUE(gradientMatrix, om) *= (om == target) - MATRIX_VALUE1(gradientMatrix, om) - inner;
}

number costFunction(Feature *input, uint8 target) {
    //Aux variables
    Matrix *inputMatrix = FEATURE_GETMATRIX(input, 0);
    return (-log(MATRIX_VALUE(FEATURE_GETMATRIX(input, 0), 0, target))/inputMatrix->m);
}
