#include "hip/hip_runtime.h"
#include "lenet.h"

/* ----- FORWARD FUNCTIONS ----- */
void activation_forward(Feature *output, Array *bias, number (*action)(number)) {
    uint on, om, matrixSize;
    Matrix *outputMatrix;
    for(on = 0; on < output->n; on++) {
        outputMatrix = FEATURE_GETMATRIX(output, on);
        matrixSize = MATRIX_SIZE(outputMatrix);
        for(om = 0; om < matrixSize; om++)
            MATRIX_VALUE1(outputMatrix, om) = action(MATRIX_VALUE1(outputMatrix, om) + ARRAY_VALUE(bias, on));
    }
}

__global__ void convolute_forward(Matrix *input, Matrix *weight, Matrix *output) {
    uint tn = threadIdx.y, tm = threadIdx.x;
    uint tid = tn*blockDim.x + tm;
    //Shared memory
    __shared__ number s_input[blockDim.y][blockDim.x];
    __shared__ number s_weight[weight->n][weight->m];
    //Load shared mem from global mem
    s_input[tn][tm] = MATRIX_VALUE(input, tn, tm);
    if(tid < (weight->n*weight->m))
        s_weight[tn][tm] = MATRIX_VALUE(weight, tn, tm);
    __syncthreads();
    //Thread id inside output matrix
    if(tid < (output->n*output->m)) {
        //Aux variables
        uint wn, wm;
        number result = 0;
        //Weight matrix loop - KERNEL
        for(wn = 0; wn < weight->n; wn++)
        for(wm = 0; wm < weight->m; wm++)
            result += s_input[(tn + wn), (tm + wm)]) * s_weight[wn, wm];
        //Write back result
        atomicAdd(MATRIX_POINTER(output, tn, tm), result);
    }
}

__global__ void convolution_forward(Feature **input, LeNet lenet) {
    uint bn = blockIdx.y,  bm = blockIdx.x;
    uint tn = threadIdx.y, tm = threadIdx.x;
    uint tid = tn*blockDim.x + tm;
    //Aux variables
    Matrix *input  = FEATURE_GETMATRIX(*input, bn),
           *output = FEATURE_GETMATRIX(*(input + 1), bm),
           *weight = WEIGHT_GETMATRIX(lenet.weight, bn, bm);
    // ---- Convolution ---- //
    //Shared memory
    __shared__ number s_input[blockDim.y][blockDim.x];
    __shared__ number s_weight[weight->n][weight->m];
    //Load shared mem from global mem
    s_input[tn][tm] = MATRIX_VALUE(input, tn, tm);
    if(tid < (weight->n*weight->m))
        s_weight[tn][tm] = MATRIX_VALUE(weight, tn, tm);
    __syncthreads();
    //Thread id inside output matrix
    if(tid < (output->n*output->m)) {
        //Aux variables
        uint wn, wm;
        number result = 0;
        //Weight matrix loop - KERNEL
        for(wn = 0; wn < weight->n; wn++)
        for(wm = 0; wm < weight->m; wm++)
            result += s_input[(tn + wn), (tm + wm)]) * s_weight[wn, wm];
        //Write back result
        atomicAdd(MATRIX_POINTER(output, tn, tm), result);
    }

    //Activation function
    //activation_forward(output, lenet.bias, ReLU);
}

void subsampling_forward(Feature **input) {
    Feature *output = *(input + 1);
    //Aux variables
    Matrix *inputMatrix, *outputMatrix;
    uint o, on, om, ln, lm, aux_n, aux_m;
    number max, aux;
    const uint ln_length = FEATURE_GETMATRIX(*input, 0)->n / FEATURE_GETMATRIX(output, 0)->n,
               lm_length = FEATURE_GETMATRIX(*input, 0)->m / FEATURE_GETMATRIX(output, 0)->m;
    //Ouput array loop
    for(o = 0; o < output->n; o++) {
        inputMatrix = FEATURE_GETMATRIX(*input, o);
        outputMatrix = FEATURE_GETMATRIX(output, o);
        //Output matrix loop
        for(on = 0; on < outputMatrix->n; on++)
        for(om = 0; om < outputMatrix->m; om++) {
            //Subsampling
            max = -1, aux_n = ln_length*on, aux_m = lm_length*om;
            for(ln = 0; ln < ln_length; ln++)
            for(lm = 0; lm < lm_length; lm++) {
                aux = MATRIX_VALUE(inputMatrix, (aux_n + ln), (aux_m + lm));
                max = (aux > max) ? aux:max;
            }
            MATRIX_VALUE(outputMatrix, on, om) = max;
        }
    }
}

void dotproduct_forward(Feature **input, LeNet lenet) {
    Feature *output = *(input + 1);
    //Aux variables
    uint wn1, wn2, wm, wn1_aux;
    Matrix *inputMatrix, 
           *weightMatrix = WEIGHT_GETMATRIX1(lenet.weight, 0),
           *outputMatrix = FEATURE_GETMATRIX(output, 0);
    const uint wn1_length = (*input)->n, wn2_length = (weightMatrix->n)/wn1_length;
    //Dot product
    for(wn1 = 0; wn1 < wn1_length; wn1++) {
        inputMatrix = FEATURE_GETMATRIX(*input, wn1);
        wn1_aux = wn1*wn2_length;
        for(wn2 = 0; wn2 < wn2_length; wn2++)
        for(wm = 0; wm < weightMatrix->m; wm++)
            MATRIX_VALUE1(outputMatrix, wm) += MATRIX_VALUE1(inputMatrix, wn2) * MATRIX_VALUE(weightMatrix, (wn1_aux + wn2), wm);
    }
    //Activation function
    for(wm = 0; wm < lenet.bias->n; wm++)
        MATRIX_VALUE1(outputMatrix, wm) = ReLU(MATRIX_VALUE1(outputMatrix, wm) + ARRAY_VALUE(lenet.bias, wm));
}
