//srun ./lenet -N1 --gres=gpu:1
#include "lenet5/lenet.h"
#define LENET_FILE "modelfnt.dat"

__host__ uint testing(LeNet *lenet, uint8 testImage[][IMG_SIZE], uint8 *testLabel, uint totalSize) {
    printf("--------\n");
    printf("TESTING\n");
    //Aux variables
    const uint INPUT_SIZE = F_PARALLEL*IMG_SIZE*sizeof(uint8);
    const uint LABEL_SIZE = F_PARALLEL*sizeof(uint8);
    uint i, rightPredictions = 0;
    //Loop
    #pragma omp parallel for
    for (i = 0; i < totalSize; i+= F_PARALLEL) {
        uint8 *d_input, *d_label;
        //Copy input to device
        hipMalloc((void **)&d_input, INPUT_SIZE);
        hipMemcpy(d_input, testImage[i], INPUT_SIZE, hipMemcpyHostToDevice);
        //Copy labels to device
        hipMalloc((void **)&d_label, LABEL_SIZE);
        hipMemcpy(d_label, &testLabel[i], LABEL_SIZE, hipMemcpyHostToDevice);
        //Predict
        #pragma omp critical
        rightPredictions += predict(lenet, d_input, d_label);
        //Free
        hipFree(d_input);
        hipFree(d_label);
    }
    return rightPredictions;
}

__host__ void training(LeNet *lenet, uint8 trainImage[][IMG_SIZE], uint8 *trainLabel, const uint totalSize) {
    printf("--------\n");
    printf("TRAINING\n");
    //Aux variables
    const uint INPUT_SIZE = B_PARALLEL*IMG_SIZE*sizeof(uint8);
    const uint LABEL_SIZE = B_PARALLEL*sizeof(uint8);
    uint i;
    uint8 *d_input, *d_label;
    for (i = 0; i < totalSize; i += B_PARALLEL) {
        //Copy input to device
        hipMalloc((void **)&d_input, INPUT_SIZE);
        hipMemcpy(d_input, trainImage[i], INPUT_SIZE, hipMemcpyHostToDevice);
        //Copy labels to device
        hipMalloc((void **)&d_label, LABEL_SIZE);
        hipMemcpy(d_label, &trainLabel[i], LABEL_SIZE, hipMemcpyHostToDevice);
        //Train
        trainBatch(lenet, d_input, d_label);
        //Free
        hipFree(d_input); hipFree(d_label);
    }
}

__host__ void load(LeNet *h_lenet, LeNet *d_lenet, char filename[]) {
    //Read file
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        printf("Model not found \n");
        exit(0);
    }
    fread(h_lenet, sizeof(LeNet), 1, fp);
    fclose(fp);
    //Copy info to device
    hipMemcpy(d_lenet, h_lenet, sizeof(LeNet), hipMemcpyHostToDevice);
}

int main() {
    printf("-------------------\n");
    printf("PROCESS STARTED\n");
    //Events
    hipEvent_t start, stop_training, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop_training); hipEventCreate(&stop);
    //Allocate memory
    LeNet *h_lenet, *d_lenet;
    hipHostMalloc((void **)&h_lenet, sizeof(LeNet));
    hipMalloc((void **)&d_lenet, sizeof(LeNet));
    //Training
    bool train = true;
    //Testing data
    static uint8 testImage[NUM_TEST][IMG_SIZE]; 
    static uint8 testLabel[NUM_TEST];
    load_testData(testImage, testLabel);
    //Process starts
    hipEventRecord(start);
    if(train) {
        //Train data
        static uint8 trainImage[NUM_TRAIN][IMG_SIZE];
        static uint8 trainLabel[NUM_TRAIN];
        load_trainingData(trainImage, trainLabel);
        //Initial
        load(h_lenet, d_lenet, (char *)LENET_FILE); //setInitialValues(lenet);
        hipEventRecord(start);
        training(d_lenet, trainImage, trainLabel, NUM_TRAIN);
    }
    else {
        //Testing
        load(h_lenet, d_lenet, (char *)LENET_FILE);
        hipEventRecord(start);
    }
    hipEventRecord(stop_training);
    //Testing
    uint rightPredictions = testing(d_lenet, testImage, testLabel, NUM_TEST);
    hipEventRecord(stop);
    //Process ends
    hipEventSynchronize(stop_training);
    hipEventSynchronize(stop);
    float exec_time = 0, t_time = 0;
    hipEventElapsedTime(&t_time, start, stop_training);
    hipEventElapsedTime(&exec_time, start, stop);
    printf("-------------------\n");
    printf("PROCESS FINISHED\n ");
    printf("Results: %d/%d\n", rightPredictions, NUM_TEST);
    printf("Training time (s): %f \n", t_time/1000);
    printf("Execution time (s): %f \n", exec_time/1000);
    //Memory free
    printf("-------------------\n");
    printf("FREE LENET MEMORY\n");
    hipFree(h_lenet); hipFree(d_lenet);
    hipDeviceReset();
    return 0;
}

